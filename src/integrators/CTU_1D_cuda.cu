
#include <hip/hip_runtime.h>
/*! \file CTU_1D_cuda.cu
 *  \brief Definitions of the cuda CTU algorithm functions. */

#ifdef CUDA

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "../utils/gpu.hpp"
#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../hydro/hydro_cuda.h"
#include "../integrators/CTU_1D_cuda.h"
#include "../reconstruction/pcm_cuda.h"
#include "../reconstruction/plmp_cuda.h"
#include "../reconstruction/plmc_cuda.h"
#include "../reconstruction/ppmp_cuda.h"
#include "../reconstruction/ppmc_cuda.h"
#include "../riemann_solvers/exact_cuda.h"
#include "../riemann_solvers/roe_cuda.h"
#include "../riemann_solvers/hllc_cuda.h"
#include "../cooling/cooling_cuda.h"
#include "../utils/error_handling.h"
#include "../io/io.h"



void CTU_Algorithm_1D_CUDA(Real *host_conserved0, Real *host_conserved1, Real *d_conserved, int nx, int x_off, int n_ghost, Real dx, Real xbound, Real dt, int n_fields)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //host_conserved0 contains the values at time n
  //host_conserved1 will contain the values at time n+1

  // Initialize dt values

  int n_cells = nx;
  int ny = 1;
  int nz = 1;

  // set the dimensions of the cuda grid
  ngrid = (n_cells + TPB - 1) / TPB;
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  if ( !memory_allocated ) {

    // allocate an array on the CPU to hold max_dti returned from each thread block
    CudaSafeCall( cudaHostAlloc(&host_dti_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaHostAlloc(&host_dt_array, ngrid*sizeof(Real), cudaHostAllocDefault) );
    #endif

    // allocate memory on the GPU
    dev_conserved = d_conserved;
    //CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,   (n_fields)*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #if defined COOLING_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif

    #ifndef DYNAMIC_GPU_ALLOC
    // If memory is single allocated: memory_allocated becomes true and succesive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;
    #endif
  }

  // copy the conserved variable array onto the GPU
  #ifndef HYDRO_GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, host_conserved0, n_fields*n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
  CudaCheckError();
  #endif // HYDRO_GPU


  // Step 1: Do the reconstruction
  #ifdef PCM
  hipLaunchKernelGGL(PCM_Reconstruction_1D, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif
  #ifdef PLMP
  hipLaunchKernelGGL(PLMP_cuda, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PLMC
  hipLaunchKernelGGL(PLMC_cuda, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PPMP
  hipLaunchKernelGGL(PPMP_cuda, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif
  #ifdef PPMC
  hipLaunchKernelGGL(PPMC_cuda, dimGrid, dimBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  CudaCheckError();
  #endif


  // Step 2: Calculate the fluxes
  #ifdef EXACT
  hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef ROE
  hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef HLLC
  hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dimGrid, dimBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  CudaCheckError();

  #ifdef DE
  // Compute the divergence of Vel before updating the conserved array, this solves syncronization issues when adding this term on Update_Conserved_Variables
  hipLaunchKernelGGL(Partial_Update_Advected_Internal_Energy_1D, dimGrid, dimBlock, 0, 0,  dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, dx, dt, gama, n_fields );
  #endif


  // Step 3: Update the conserved variable array
  hipLaunchKernelGGL(Update_Conserved_Variables_1D, dimGrid, dimBlock, 0, 0, dev_conserved, F_x, n_cells, x_off, n_ghost, dx, xbound, dt, gama, n_fields);
  CudaCheckError();


  // Sychronize the total and internal energy, if using dual-energy formalism
  #ifdef DE
  hipLaunchKernelGGL(Select_Internal_Energy_1D, dimGrid, dimBlock, 0, 0, dev_conserved, nx, n_ghost, n_fields);
  hipLaunchKernelGGL(Sync_Energies_1D, dimGrid, dimBlock, 0, 0, dev_conserved, n_cells, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif

  #ifdef DYNAMIC_GPU_ALLOC
  // If memory is not single allocated then free the memory every timestep.
  Free_Memory_CTU_1D();
  #endif

  return;


}

void Free_Memory_CTU_1D() {

  // free the CPU memory
  CudaSafeCall( cudaFreeHost(host_dti_array) );
  #if defined COOLING_GPU
  CudaSafeCall( cudaFreeHost(host_dt_array) );
  #endif

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(F_x);
  cudaFree(dev_dti_array);
  #if defined COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

}


#endif //CUDA
